#include "hip/hip_runtime.h"
/*
#######################################################
#                                                     #
#	Final Porject, Programación Multinúcleo           #
#	Daniel Monzalvo, Miguel del Moral                 #
#													  #
#   Rainbow table construction in parallel,           #
#   Hash algorythm sequecial                          #
#                                                     #
#######################################################
*/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Limits of char values
#define maxChar '~'
#define minChar ' '

#define SIZE_MD5 16

// number of bytes to be processed by GPU
#define NUM_B 1000
#define NUM_B_B 1000
#define NUM_T_B 1024

char* nextChar(char* str){
	int len = strlen(str);
	char* s2 = (char*) malloc (len);

	for(int i = len-1; i>=0; --i){
		s2[i] = str[i];
	}

	for (int i = len-1; i >= 0; --i){
		//s2[i] = str[i];
		if(s2[i] < maxChar-1){
			s2[i] =  s2[i] + 1;
			//std::cout << " to: " <<s2[i] << "\n";
			return s2;
		}
		else{
			s2[i] = minChar;
		}
	}
	//std::cout << "cambio\n";
	s2 = (char*) realloc (s2, len+1);
	//s2[0] = 0;
	char temp = s2[0];
	char temp2;
	for (int i = 1; i < len+1; ++i){
		temp2 = s2[i];
		s2[i] = temp;
		temp = temp2;
	}
	//std::cout << "changed\n";
	s2[0] = minChar;
	return s2;
}


char** makeBrick(char** array, char* s, int lh){
	//std::cout << s << " ini\n";
	int cont = 0;
	
	while (cont < (NUM_B/lh) && strlen(s) <= lh){
		array[cont] = s;
		//std::cout << cont << ": -" << array[cont] << "-\n";
		++cont;
		s = nextChar(s);
	}
	while (cont < (NUM_B/lh)){
		array[cont] = "";
		++cont;
	}
	return array;
}

void print(char** arr, int lh){
	for(int i = 0; i < NUM_B/lh; ++i){
		//if(arr[i] == "")
			//break;
		std::cout << i << ": -" << arr[i] << "-\n";
	}
}

__global__ void hashBrick(char** words, char** hashes){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	hashes[id] = "a";
}


int main(int argc, char* argv[]){

	int ll, lh, al, blocks, threads;

	if(argc < 3){
		std::cout << "please choose algorythm: (1)MD5 (2)SHA, and length." << std::endl;
		return 0;
	}
	else if(argc > 4){
		std::cout << "too many arguments.\n";
		return 0;
	}
	else if(argc == 4){
		if (argv[2] > argv[3]){
			std::cout << "Lower limit higher than hig limit.\n";
			return 0;
		}
		if (argv[2] <= 0){
			std::cout << "Lower limit too low.\n";
			return 0;
		}
		ll = atoi(argv[2]);
		lh = atoi(argv[3]);
	}
	else{
		ll = atoi(argv[2]);
		lh = ll;
	}

	float tiempo1;
	hipEvent_t inicio, fin;

	hipEventCreate(&inicio);
	hipEventCreate(&fin);
	hipEventRecord( inicio, 0);
	

	al = 0;
	int it = ll;
	while(it <= lh){
		al += pow(94,it);
		it++;
	}

	blocks = NUM_B/NUM_B_B;
	if(blocks < NUM_B*NUM_B_B)
		blocks++;

	threads = (NUM_B/lh)/blocks;

	std::cout << "Words = " << (NUM_B/lh) << "\n";
	std::cout << "Total = " << al << "\n";

	int loops = al / (NUM_B/lh);
	if(loops * (NUM_B/lh) < al)
		loops++;
	std::cout << "Loops = " << loops << "\n";

	char* first = (char*) malloc (lh);
	for (int i = lh-1; i >= ll; --i){
		first[i] = minChar;
	}

	//std::cout << first << " first\n";

	// Declare arrays

	// CPU word aray
	char** arr = (char**) malloc (sizeof(char*)*(NUM_B/lh));
	for(int i = 0; i < (NUM_B/lh); ++i){
		arr[i] = (char*)malloc(lh);
		//std::cout << "Alloc with 0 = " << (int)arr[i][0] << "\n";
	}
	//CPU hash array
	char** hash = (char**) malloc (sizeof(char*)*(NUM_B/lh));
	//GPU word array
	char** arr_dev;
	char** h_temp = (char**) malloc (sizeof(char*)*(NUM_B/lh));
	for (int i  =0; i < (NUM_B/lh) ; ++i){
		hipMalloc((void**)&(h_temp[i]), lh);
	}
	hipMalloc( (void**)&arr_dev, sizeof(char*)*(NUM_B/lh));
	hipMemcpy(arr_dev, h_temp, sizeof(char*)*(NUM_B/lh), hipMemcpyHostToDevice);
	// GPU hash array
	char** hash_dev;
	char** h_temp2 = (char**) malloc (sizeof(char*)*(NUM_B/lh));
	for (int i  =0; i < (NUM_B/lh) ; ++i){
		hipMalloc((void**)&h_temp2[i], lh);
	}
	hipMalloc( (void**)&hash_dev, sizeof(char*)*(NUM_B/lh));
	hipMemcpy(hash_dev, h_temp2, sizeof(char*)*(NUM_B/lh), hipMemcpyHostToDevice);
	
	for (int i = 0; i < loops; ++i){
		std::cout << "entro\n";
		arr = makeBrick(arr, first, lh);

		for (int j  =0; j < (NUM_B/lh); ++j){
			hipMemcpy(h_temp[j], arr[j], lh, hipMemcpyHostToDevice);
		}

		hashBrick<<<blocks,threads>>>(arr_dev, hash_dev);
		hipDeviceSynchronize();
		first = nextChar(arr[(NUM_B/lh)-1]);
		char** res = (char**)malloc((NUM_B/lh)*sizeof(char*));
		hipMemcpy(res, hash_dev, (NUM_B/lh)*sizeof(char*), hipMemcpyDeviceToHost);
		for (int j = 0; j < (NUM_B/lh);++j){
			hipMemcpy(hash[j], res[j], lh, hipMemcpyDeviceToHost);
		}
		
		print(hash, lh);
	}

	hipEventRecord(fin, 0);
	hipEventSynchronize(fin);
	hipEventElapsedTime(&tiempo1, inicio, fin);

	std::cout << "Time: " << tiempo1 << std::endl;
	
	//free(arr);

	return 0;
}