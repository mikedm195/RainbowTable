#include "hip/hip_runtime.h"
/*
#######################################################
#                                                     #
#	Final Porject, Programación Multinúcleo           #
#	Daniel Monzalvo, Miguel del Moral                 #
#													  #
#   Rainbow table construction in parallel,           #
#   Hash algorythm sequecial                          #
#                                                     #
#######################################################
*/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>

// Limits of char values
#define maxChar '~'
#define minChar ' '

#define SIZE_MD5 17
#define SIZE_SHA 33

// number of bytes to be processed by GPU
#define NUM_B 10000
#define NUM_B_B 1000
#define NUM_T_B 1024

void nextChar(char* &str, int h){
	int len = strlen(str);

	for (int i = len-1; i >= 0; --i){
		if(str[i] < maxChar-1){
			str[i] += 1;
			return;
		}
		else{
			str[i] = minChar;
		}
	}


	if(len == h){
		str = (char*) realloc (str, len+2);
		//std::cout << "Big\n";
		len++;
	}

	char temp = str[0];
	char temp2;
	for (int i = 1; i < len+2; ++i){
		temp2 = str[i];
		str[i] = temp;
		temp = temp2;
	}
	
	str[0] = minChar;
	//std::cout << "changed to: -" << str[1] << "-\n";
}


__global__ void hashBrick(char* a, char* r, int p1, int p2, int H, int algoritmo){
	int id = threadIdx.x + (blockIdx.x * blockDim.x);
	char* word = (char*)((char*)a + (id*p1));
	char* hash = (char*)((char*)r + (id*p2));

	if(word[0] != '\0'){
		/******* AQUI VA LA LLAMADA A FUNCION DE HASHEO  *******/

		if(algoritmo == 1){
			/**** MD5 *****/
			//hash[0] = 48 + algoritmo;
			//hash[1] = '\0';

			hash[SIZE_MD5-1] = '\0';
		}
		else{
			/***** SHA *****/

			//hash[0] = 48 + algoritmo;
			//hash[1] = '\0';
			hash[SIZE_SHA-1] = '\0';
		}
	}
	else{
		hash[0] = '0';
		hash[1] = '\0';
	}

}


int main(int argc, const char* argv[]){

	int ll, al, blocks, threads, algo;

	if(argc < 3){
		std::cout << "please choose algorythm: (1)MD5 (2)SHA, and length." << std::endl;
		return 0;
	}
	else if(argc > 4){
		std::cout << "too many arguments.\n";
		return 0;
	}
	else if(argc == 4){
		if (argv[2] > argv[3]){
			std::cout << "Lower limit higher than hig limit.\n";
			return 0;
		}
		if (argv[2] <= 0){
			std::cout << "Lower limit too low.\n";
			return 0;
		}
		ll = atoi(argv[2]);
		algo = atoi(argv[1]);
	}
	else{
		ll = atoi(argv[2]);
		algo = atoi(argv[1]);
	}

	const int lh = (argc == 4)? atoi(argv[3]) : atoi(argv[2]);

	float tiempo1;
	hipEvent_t inicio, fin;

	hipEventCreate(&inicio);
	hipEventCreate(&fin);
	hipEventRecord( inicio, 0);
	

	al = 0;
	int it = ll;
	while(it <= lh){
		al += pow(94,it);
		it++;
	}

	blocks = NUM_B/NUM_B_B;
	if(blocks < NUM_B*NUM_B_B)
		blocks++;

	threads = (NUM_B/lh)/blocks;

	std::cout << "Words = " << (NUM_B/lh) << "\n";
	std::cout << "Total = " << al << "\n";

	int loops = al / (NUM_B/lh);
	if(loops * (NUM_B/lh) < al)
		loops++;
	std::cout << "Loops = " << loops << "\n";

	char* first = (char*) malloc (lh+1);
	for (int i = 0; i < ll; ++i){
		first[i] = minChar;
	}
	first[ll] = '\0';

	//std::cout << first << " first\n";

	// Declare arrays
	const int width = NUM_B/lh;
	const int height = lh+1;
	const int height2 = (algo==1)? SIZE_MD5 : SIZE_SHA;
	size_t host_pitch1 = height*sizeof(char);
	size_t host_pitch2 = height2*sizeof(char);

	// CPU word aray
	char arr[width][height];

	//CPU hash array
	char hash[width][height2];

	//GPU word array
	char* arr_dev;
	size_t pitch1;
	hipMallocPitch((void**)&arr_dev, &pitch1, height, width);
	// GPU hash array
	char* hash_dev;
	size_t pitch2;
	hipMallocPitch((void**)&hash_dev, &pitch2, height2, width);

	std::ofstream f;
	f.open("Table.txt");
	
	for (int i = 0; i < loops; ++i){

		for(int j = 0; j < width; ++j){
			if(strlen(first) <= height-1){
				for(int k = 0; k < height; ++k){
					arr[j][k] = first[k];
				}
				nextChar(first, lh);
			}
			else{
				//std::cout << "nop: " << strlen(first) << " > "  << height-1 << std::endl;
				arr[j][0] = '\0';
			}
		}

		hipMemcpy2D(arr_dev, pitch1, arr, host_pitch1, height*sizeof(char), width, hipMemcpyHostToDevice);

		hashBrick<<<blocks,threads>>>(arr_dev, hash_dev, pitch1, pitch2, height, algo);
		hipDeviceSynchronize();

		hipMemcpy2D(hash, host_pitch2, hash_dev, pitch2, height2*sizeof(char), width, hipMemcpyDeviceToHost);

		for(int j = 0; j < width; ++j){
			if(strlen(arr[j])>0)
				f << arr[j] << '\t' << hash[j] << '\n';
		}
	}

	f.close();

	hipEventRecord(fin, 0);
	hipEventSynchronize(fin);
	hipEventElapsedTime(&tiempo1, inicio, fin);

	std::cout << "Time: " << tiempo1 << std::endl;
	
	//free(arr);

	return 0;
}