#include "hip/hip_runtime.h"
/*
#######################################################
#                                                     #
#	Final Porject, Programación Multinúcleo           #
#	Daniel Monzalvo, Miguel del Moral                 #
#													  #
#   Rainbow table construction in parallel,           #
#   Hash algorythm sequecial                          #
#                                                     #
#######################################################
*/

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <fstream>
//#include "md5.h"
//#include "sha.h"
// Limits of char values
#define maxChar '~'
#define minChar ' '

#define SIZE_MD5 33
#define SIZE_SHA 33

// number of bytes to be processed by GPU
#define NUM_B 10000
#define NUM_B_B 1000
#define NUM_T_B 1024


// *************************** MD5 ************************//

__device__ unsigned func0( unsigned abcd[] ){
    return ( abcd[1] & abcd[2]) | (~abcd[1] & abcd[3]);}

__device__ unsigned func1( unsigned abcd[] ){
    return ( abcd[3] & abcd[1]) | (~abcd[3] & abcd[2]);}

__device__ unsigned func2( unsigned abcd[] ){
    return  abcd[1] ^ abcd[2] ^ abcd[3];}

__device__ unsigned func3( unsigned abcd[] ){
    return abcd[2] ^ (abcd[1] |~ abcd[3]);}

typedef unsigned (*DgstFctn)(unsigned a[]);

typedef union uwb {
    unsigned w;
    unsigned char b[4];
} MD5union;

typedef unsigned DigestArray[4];

__device__ unsigned rol( unsigned r, short N )
{
    unsigned  mask1 = (1<<N) -1;
    return ((r>>(32-N)) & mask1) | ((r<<N) & ~mask1);
}

__device__ unsigned *calctable( unsigned *k)
{
    double s, pwr = 2;
    int i;
    for (int j = 1; i < 32; ++i){
    	pwr *= 2;
    }
    //pwr = pow( 2, 32);
    for (i=0; i<64; i++) {
        s = fabs(sin((double)(1+i)));
        k[i] = (unsigned)( s * pwr );
    }
    return k;
}

__device__ unsigned *getMd5( const char *msg, int mlen){
	DigestArray h0 = { 0x67452301, 0xEFCDAB89, 0x98BADCFE, 0x10325476 };
    DgstFctn ff[] = { &func0, &func1, &func2, &func3 };
    short M[] = { 1, 5, 3, 7 };
    short O[] = { 0, 1, 5, 0 };
    short rot0[] = { 7,12,17,22};
    short rot1[] = { 5, 9,14,20};
    short rot2[] = { 4,11,16,23};
    short rot3[] = { 6,10,15,21};
    short *rots[] = {rot0, rot1, rot2, rot3 };
    unsigned kspace[64];
    unsigned *k;

    DigestArray h;
    DigestArray abcd;
    DgstFctn fctn;
    short m, o, g;
    unsigned f;
    short *rotn;
    union {
        unsigned w[16];
        char     b[64];
    }mm;
    int os = 0;
    int grp, grps, q, p;
    unsigned char *msg2;

    if (k==NULL) k= calctable(kspace);

    for (q=0; q<4; q++) h[q] = h0[q];   // initialize

    {
        grps  = 1 + (mlen+8)/64;
        msg2 = (unsigned char*)malloc( 64*grps);
        memcpy( msg2, msg, mlen);
        msg2[mlen] = (unsigned char)0x80;
        q = mlen + 1;
        while (q < 64*grps){ msg2[q] = 0; q++ ; }
        {
            MD5union u;
            u.w = 8*mlen;
            q -= 8;
            memcpy(msg2+q, &u.w, 4 );
        }
    }

    for (grp=0; grp<grps; grp++)
    {
        memcpy( mm.b, msg2+os, 64);
        for(q=0;q<4;q++) abcd[q] = h[q];
        for (p = 0; p<4; p++) {
            fctn = ff[p];
            rotn = rots[p];
            m = M[p]; o= O[p];
            for (q=0; q<16; q++) {
                g = (m*q + o) % 16;
                f = abcd[1] + rol( abcd[0]+ fctn(abcd) + k[q+16*p] + mm.w[g], rotn[q%4]);

                abcd[0] = abcd[3];
                abcd[3] = abcd[2];
                abcd[2] = abcd[1];
                abcd[1] = f;
            }
        }
        for (p=0; p<4; p++)
            h[p] += abcd[p];
        os += 64;
    }
    return h;
}


__device__ char * md5(char* msg, int H){
    int j,k;
    char * res = "";
    int sizef = 0;
    for(int i = 0; i < H; ++i){
    	if(msg[i] == '\0')
    		break;
    	sizef++;
    }
    unsigned *d = getMd5(msg, sizef);
    MD5union u;
    char temp[33];
    char mask = 240;
    char mask2 = 15;
    temp[32] = '\0';
    char* temp2;
    int cont = 0;
    for (j=0;j<4; j++){
        u.w = d[j];
        for (k=0;k<4;k++){
            int sum = 0;
            int mult = 8;
            for (int i = 7 ; i >=4 ; i--) {
                if((u.b[k] & (1 << i)) != 0 )
                    sum+=mult;
                // printf("%d",(u.b[k] & (1 << i)) != 0 );
                mult/=2;
            }
            // printf("\n%d\n",sum );
            if(sum<=9)
                temp[cont++] = (char)sum+48;
            else
                temp[cont++] = (char)sum+87;
            sum=0;
            mult = 8;
            for (int i = 3 ; i >=0 ; i--) {
                if((u.b[k] & (1 << i)) != 0 )
                    sum+=mult;
                //  printf("%d",(u.b[k] & (1 << i)) != 0 );
                mult/=2;
            }
            if(sum<=9)
                temp[cont++] = (char)sum+48;
            else
                temp[cont++] = (char)sum+87;
            // printf("%02x", u.b[k] );
        }
    }
    // printf("\n");
    // printf("%s\n",temp );
    return temp;
}


//***************************************  MD5 - FIN *************************************//

//*************************************** crear nueva cadena ******************************//
void nextChar(char* &str, int h){
	int len = strlen(str);

	for (int i = len-1; i >= 0; --i){
		if(str[i] < maxChar-1){
			str[i] += 1;
			return;
		}
		else{
			str[i] = minChar;
		}
	}


	if(len == h){
		str = (char*) realloc (str, len+2);
		//std::cout << "Big\n";
		len++;
	}

	char temp = str[0];
	char temp2;
	for (int i = 1; i < len+2; ++i){
		temp2 = str[i];
		str[i] = temp;
		temp = temp2;
	}

	str[0] = minChar;
	//std::cout << "changed to: -" << str[1] << "-\n";
}

/******************************** KERNEL ***********************************************/
__global__ void hashBrick(char* a, char* r, int p1, int p2, int H, int algoritmo){
	int id = threadIdx.x + (blockIdx.x * blockDim.x);
	char* word = (char*)((char*)a + (id*p1));
	char* hash = (char*)((char*)r + (id*p2));

	if(word[0] != '\0'){
		/******* AQUI VA LA LLAMADA A FUNCION DE HASHEO  *******/

		if(algoritmo == 1){
			/**** MD5 *****/
			//hash[0] = 48 + algoritmo;
			//hash[1] = '\0';

			char* res = md5(word, H);

			for (int i = 0; i < SIZE_MD5-1; ++i){
				hash[i] = res[i];
			}

			hash[SIZE_MD5-1] = '\0';
		}
		else{
			/***** SHA *****/

			//hash[0] = 48 + algoritmo;
			//hash[1] = '\0';
			hash[SIZE_SHA-1] = '\0';
		}
	}
	else{
		hash[0] = '0';
		hash[1] = '\0';
	}

}


/******************* Main ***********************************/

int main(int argc, const char* argv[]){
	//md5("hola");
	//sha("hola");
	int ll, al, blocks, threads, algo;

	if(argc < 3){
		std::cout << "please choose algorythm: (1)MD5 (2)SHA, and length." << std::endl;
		return 0;
	}
	else if(argc > 4){
		std::cout << "too many arguments.\n";
		return 0;
	}
	else if(argc == 4){
		if (argv[2] > argv[3]){
			std::cout << "Lower limit higher than hig limit.\n";
			return 0;
		}
		if (argv[2] <= 0){
			std::cout << "Lower limit too low.\n";
			return 0;
		}
		ll = atoi(argv[2]);
		algo = atoi(argv[1]);
	}
	else{
		ll = atoi(argv[2]);
		algo = atoi(argv[1]);
	}

	const int lh = (argc == 4)? atoi(argv[3]) : atoi(argv[2]);

	float tiempo1;
	hipEvent_t inicio, fin;

	clock_t t;
	t = clock();

	hipEventCreate(&inicio);
	hipEventCreate(&fin);
	hipEventRecord(inicio, 0);


	al = 0;
	int it = ll;
	while(it <= lh){
		al += pow(94,it);
		it++;
	}

	// Calculo de bloques y threads
	blocks = NUM_B/NUM_B_B;
	if(blocks < NUM_B*NUM_B_B)
		blocks++;

	threads = (NUM_B/lh)/blocks;

	std::cout << "Words = " << (NUM_B/lh) << "\n";
	std::cout << "Total = " << al << "\n";

	// Calculo de bricks a procesar

	int loops = al / (NUM_B/lh);
	if(loops * (NUM_B/lh) < al)
		loops++;
	std::cout << "Loops = " << loops << "\n";

	char* first = (char*) malloc (lh+1);
	for (int i = 0; i < ll; ++i){
		first[i] = minChar;
	}
	first[ll] = '\0';

	//std::cout << first << " first\n";

	// Declare arrays
	const int width = NUM_B/lh;
	const int height = lh+1;
	const int height2 = (algo==1)? SIZE_MD5 : SIZE_SHA;
	size_t host_pitch1 = height*sizeof(char);
	size_t host_pitch2 = height2*sizeof(char);

	// CPU word aray
	char arr[width][height];

	//CPU hash array
	char hash[width][height2];

	//GPU word array
	char* arr_dev;
	size_t pitch1;
	hipMallocPitch((void**)&arr_dev, &pitch1, height, width);
	// GPU hash array
	char* hash_dev;
	size_t pitch2;
	hipMallocPitch((void**)&hash_dev, &pitch2, height2, width);

	// Archivo
	std::ofstream f;
	f.open("Table.txt");

	// Recorrer todos los bricks
	for (int i = 0; i < loops; ++i){

		// Crear Bricks
		for(int j = 0; j < width; ++j){
			if(strlen(first) <= height-1){
				for(int k = 0; k < height; ++k){
					arr[j][k] = first[k];
				}
				nextChar(first, lh);
			}
			else{
				//std::cout << "nop: " << strlen(first) << " > "  << height-1 << std::endl;
				arr[j][0] = '\0';
			}
		}

		// Copiar a Tarjeta
		hipMemcpy2D(arr_dev, pitch1, arr, host_pitch1, height*sizeof(char), width, hipMemcpyHostToDevice);

		// Procesar el Kernel
		hashBrick<<<blocks,threads>>>(arr_dev, hash_dev, pitch1, pitch2, height, algo);
		hipDeviceSynchronize();

		// Copiar a RAM
		hipMemcpy2D(hash, host_pitch2, hash_dev, pitch2, height2*sizeof(char), width, hipMemcpyDeviceToHost);

		// Copiar a DISCO
		for(int j = 0; j < width; ++j){
			if(strlen(arr[j])>0)
				f << arr[j] << '\t' << hash[j] << '\n';
		}
	}

	f.close();

	hipEventRecord(fin, 0);
	hipEventSynchronize(fin);
	hipEventElapsedTime(&tiempo1, inicio, fin);

	t = clock()-t; 

	std::cout << "Time: " << (((float)t) / CLOCKS_PER_SEC) << std::endl;

	//free(arr);

	return 0;
}
